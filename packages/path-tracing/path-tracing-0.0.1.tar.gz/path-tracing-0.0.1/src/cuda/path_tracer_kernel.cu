#include "hip/hip_runtime.h"
#include "path_tracer_kernel.cuh"

#include "camera.h"
#include "scene.h"

#include <algorithm>
#include <cstdio>

namespace cuda {

__device__ static Vec3f radiance(Ray ray, const Scene &scene, RandomGenerator &rand) {
    constexpr int MAX_DEPTH = 5;
    int depth = 0;
    Vec3f factor = Vec3f::One();
    Vec3f color = Vec3f::Zero();
    while (true) {
        assert(is_close(ray.dir.norm(), 1));
        Hit hit;
        scene.intersect(ray, hit);
        if (!hit.is_hit()) {
            color += factor.cwiseProduct(scene.background);
            break;
        }
        color += factor.cwiseProduct(hit.material->emit);
        if (depth >= MAX_DEPTH) {
            break;
        }
        Scatter scat = hit.material->scatter(ray, hit, rand);
        if (scat.color == Vec3f::Zero()) {
            break;
        }
        factor = factor.cwiseProduct(scat.color);
        ray = scat.ray;
        depth++;
    }
    return color;
}

static constexpr int BLOCK_SIZE = 32;

__global__ static void path_tracer_kernel(Image image, Camera camera, int num_samples, Scene scene) {
    //    if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0) {
    //        for (int i = 0; i < meshes.size; i++) {
    //            auto &mesh = meshes[i];
    //            for (int k = 0; k < mesh.bvh.inner_nodes.size; k++) {
    //                auto &bbox = mesh.bvh.inner_nodes[k].bbox;
    //                printf("GLOBAL MIN (%f %f %f) MAX (%f %f %f)\n", bbox.minv.x, bbox.minv.y, bbox.minv.z,
    //                bbox.maxv.x,
    //                       bbox.maxv.y, bbox.maxv.z);
    //            }
    //            for (int k = 0; k < mesh.bvh.leaf_nodes.size; k++) {
    //                auto &bbox = mesh.bvh.leaf_nodes[k].bbox;
    //                printf("GLOBAL MIN (%f %f %f) MAX (%f %f %f)\n", bbox.minv.x, bbox.minv.y, bbox.minv.z,
    //                bbox.maxv.x,
    //                       bbox.maxv.y, bbox.maxv.z);
    //            }
    //        }
    //    }
    int x = (int)(blockIdx.x * BLOCK_SIZE + threadIdx.x);
    int y = (int)blockIdx.y;
    if (x >= camera.width) {
        return;
    }
    int idx = y * camera.width + x;

    num_samples /= 4;

    Vec3f color = Vec3f::Zero();

    RandomGenerator rand(idx);

    for (int sy = 0; sy < 2; sy++) {     // 2x2 subpixel rows
        for (int sx = 0; sx < 2; sx++) { // 2x2 subpixel cols
            Vec3f r = Vec3f::Zero();
            for (int s = 0; s < num_samples; s++) {
                float r1 = 2.f * rand.next();
                float dx = r1 < 1 ? sqrt(r1) - 1 : 1 - sqrt(2 - r1);
                float r2 = 2.f * rand.next();
                float dy = r2 < 1 ? sqrt(r2) - 1 : 1 - sqrt(2 - r2);
                Ray ray =
                    camera.shoot_ray(((float)sx - .5f + dx) / 2 + (float)x, ((float)sy - .5f + dy) / 2 + (float)y);
                r += radiance(ray, scene, rand);
            }
            color += (r / (float)num_samples).clip(0, 1);
        }
    }
    color /= 4;
    color = color.clip(0, 1);

    image.data[idx] = color;
}

static inline int ceil_div(int x, int y) { return (x + y - 1) / y; }

void path_tracer_kernel_launch(const Image &kernel_image, const Camera &camera, int num_samples, const Scene &scene) {
    dim3 grid_size(ceil_div(camera.width, BLOCK_SIZE), camera.height);
    path_tracer_kernel<<<grid_size, BLOCK_SIZE>>>(kernel_image, camera, num_samples, scene);
    CHECK_CUDA(hipDeviceSynchronize());
}

} // namespace cuda
