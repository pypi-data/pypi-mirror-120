#include "hip/hip_runtime.h"
/*
   - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
   SLEPc - Scalable Library for Eigenvalue Problem Computations
   Copyright (c) 2002-2021, Universitat Politecnica de Valencia, Spain

   This file is part of SLEPc.
   SLEPc is distributed under a 2-clause BSD license (see LICENSE).
   - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -
*/
/*
   SLEPc singular value solver: "cyclic" (CUDA implementation)
*/
#include <slepc/private/svdimpl.h>
#include "../src/svd/impls/cyclic/cyclic.h"

PetscErrorCode MatMult_Cyclic_CUDA(Mat B,Vec x,Vec y)
{
  PetscErrorCode ierr;
  SVD            svd;
  SVD_CYCLIC     *cyclic;
  PetscScalar    *d_px,*d_py;
  PetscInt       m;

  PetscFunctionBegin;
  ierr = MatShellGetContext(B,(void**)&svd);CHKERRQ(ierr);
  cyclic = (SVD_CYCLIC*)svd->data;
  ierr = MatGetLocalSize(svd->A,&m,NULL);CHKERRQ(ierr);
  ierr = VecCUDAGetArrayRead(x,(const PetscScalar**)&d_px);CHKERRQ(ierr);
  ierr = VecCUDAGetArray(y,&d_py);CHKERRQ(ierr);
  ierr = VecCUDAPlaceArray(cyclic->x1,d_px);CHKERRQ(ierr);
  ierr = VecCUDAPlaceArray(cyclic->x2,d_px+m);CHKERRQ(ierr);
  ierr = VecCUDAPlaceArray(cyclic->y1,d_py);CHKERRQ(ierr);
  ierr = VecCUDAPlaceArray(cyclic->y2,d_py+m);CHKERRQ(ierr);
  ierr = MatMult(svd->A,cyclic->x2,cyclic->y1);CHKERRQ(ierr);
  ierr = MatMult(svd->AT,cyclic->x1,cyclic->y2);CHKERRQ(ierr);
  ierr = VecCUDAResetArray(cyclic->x1);CHKERRQ(ierr);
  ierr = VecCUDAResetArray(cyclic->x2);CHKERRQ(ierr);
  ierr = VecCUDAResetArray(cyclic->y1);CHKERRQ(ierr);
  ierr = VecCUDAResetArray(cyclic->y2);CHKERRQ(ierr);
  ierr = VecCUDARestoreArrayRead(x,(const PetscScalar**)&d_px);CHKERRQ(ierr);
  ierr = VecCUDARestoreArray(y,&d_py);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

